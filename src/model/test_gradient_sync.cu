#include "hip/hip_runtime.h"
#include "cuda_kernels.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// Function to initialize gradients with different values for each GPU
void initialize_gradients(float* gradients, int size, int num_gpus) {
    for (int gpu = 0; gpu < num_gpus; ++gpu) {
        for (int i = 0; i < size; ++i) {
            gradients[gpu * size + i] = (gpu + 1) * 10.0f + i;  // Different values for each GPU
        }
    }
}

// Function to verify synchronized gradients
bool verify_sync(float* gradients, int size, int num_gpus) {
    bool success = true;
    for (int i = 0; i < size; ++i) {
        float expected = 0.0f;
        for (int gpu = 0; gpu < num_gpus; ++gpu) {
            expected += (gpu + 1) * 10.0f + i;
        }
        expected /= num_gpus;
        
        if (fabs(gradients[i] - expected) > 1e-5) {
            printf("Error at index %d: expected %f, got %f\n", i, expected, gradients[i]);
            success = false;
        }
    }
    return success;
}

int main() {
    const int size = 1000;  // Size of gradients
    const int num_gpus = 4; // Number of GPUs to simulate
    
    // Allocate unified memory for gradients (accessible by all GPUs)
    float* gradients;
    hipMallocManaged(&gradients, size * num_gpus * sizeof(float));
    
    // Initialize gradients
    initialize_gradients(gradients, size, num_gpus);
    
    // Create CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    // Synchronize gradients
    hipError_t error = synchronize_gradients(gradients, size, num_gpus, stream);
    if (error != hipSuccess) {
        printf("Error during gradient synchronization: %s\n", hipGetErrorString(error));
        return 1;
    }
    
    // Wait for synchronization to complete
    hipStreamSynchronize(stream);
    
    // Verify results
    bool success = verify_sync(gradients, size, num_gpus);
    if (success) {
        printf("Gradient synchronization test passed!\n");
    } else {
        printf("Gradient synchronization test failed!\n");
    }
    
    // Cleanup
    hipStreamDestroy(stream);
    hipFree(gradients);
    
    return success ? 0 : 1;
} 